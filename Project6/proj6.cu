#include "hip/hip_runtime.h"
/* David LaMartina
 * lamartid@oregonstate.edu
 * Project 6: Monte Carlo Simulation with CUDA
 * CS475 Spr2019
 * Adapted from arrayMul.cu provided in CS475
 */

#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>           // Needed to run CUDA code

// Helpers for working with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

// Parallel constants
#ifndef BLOCKSIZE
#define BLOCKSIZE       32                  // # threads per block
#endif

#ifndef SIZE
#define SIZE            1 * 1024 * 1024     // array size
#endif

#ifndef TOLERANCE
#define TOLERANCE       0.00001f            // tolerance to relative error
#endif

// Ranges
const float XCMIN = 0.0;
const float XCMAX = 2.0;
const float YCMIN = 0.0;
const float YCMAX = 2.0;
const float RMIN =  0.5;
const float RMAX =  2.0;

// Monte Carlo helper prototypes
float Ranf( float, float );
int Ranf( int, int );
void TimeOfDaySeed();

__global__ void MonteCarlo( float* xcs, float* ycs, float* rs, float* rsum )
{
    unsigned int numItems = blockDim.x;
    unsigned int tnum     = threadIdx.x;
    unsigned int wgNum    = blockIdx.x;
    unsigned int gid      = blockIdx.x * blockDim.x * threadIdx.x;

    // Do Monte Carlo calculation
}

int
main( int argc, char* argv[] )
{
    TimeOfDaySeed();                                        // Seed random number geneator

    int dev = findCudaDevice( argc, (const char**)argv );   // Find GPU

    // Allocate host memory
    float* hXCS = new float[ SIZE ];
    float* hYCS = new float[ SIZE ];
    float*  hRS = new float[ SIZE ];
    float* hSum = new float[ SIZE / BLOCKSIZE ] ;

    // Fill random-value arrays
    for( int n = 0; n < SIZE; n++ )
    {
        hXCS[ n ] = Ranf( XCMIN, XCMAX );
        hYCS[ n ] = Ranf( YCMIN, YCMAX );
        hRS[  n ] = Ranf( RMIN,  RMAX  );
    }

    // Allocate device memory
    float *dXCS, *dYCS, *dRS, *dSum;

    dim3 dimsXCS( SIZE, 1, 1 );
    dim3 dimsYCS( SIZE, 1, 1 );
    dim3 dimsRS(  SIZE, 1, 1 );
    dim3 dimsSum( SIZE, 1, 1 );

    hipError_t status;
    status = hipMalloc( reinterpret_cast<void **>( &dXCS ), SIZE * sizeof( float ) );
        checkCudaErrors( status );
    status = hipMalloc( reinterpret_cast<void **>( &dYCS ), SIZE * sizeof( float ) );
        checkCudaErrors( status );
    status = hipMalloc( reinterpret_cast<void **>( &dRS  ), SIZE * sizeof( float ) );
        checkCudaErrors( status );
    status = hipMalloc( reinterpret_cast<void **>( &dSum ), SIZE / BLOCKSIZE * sizeof( float ) );
        checkCudaErrors( status );

    // Copy memory to device:
    status = hipMemcpy( dXCS, hXCS, SIZE * sizeof( float ), hipMemcpyHostToDevice );
        checkCudaErrors( status );
    status = hipMemcpy( dYCS, hYCS, SIZE * sizeof( float ), hipMemcpyHostToDevice );
        checkCudaErrors( status );
    status = hipMemcpy( dRS,  hRS,  SIZE * sizeof( float ), hipMemcpyHostToDevice );
        checkCudaErrors( status );

    // Set up execution parameters
    dim3 threads( BLOCKSIZE, 1, 1 );
    dim3 grid( SIZE / threads.x, 1, 1 );

    // Create and start timer
    hipDeviceSynchronize();

    // Allocate CUDA events for timing
    hipEvent_t start, stop;
    status = hipEventCreate( &start );
        checkCudaErrors( status );
    status = hipEventCreate( &stop );
        checkCudaErrors( status );

    // Record start event
    status = hipEventRecord( start, NULL );
        checkCudaErrors( status );

    // Execute kernel
    MonteCarlo <<< grid, threads >>>( dXCS, dYCS, dRS, dSum );

    // Record stop event
    status = hipEventRecord( stop, NULL );
        checkCudaErrors( status );

    // Wait for stop event to complete
    status = hipEventSynchronize( stop );
        checkCudaErrors( status );

    float msecTotal = 0.0f;
    status = hipEventElapsedTime( &msecTotal, start, stop );
        checkCudaErrors( status );

    // Compute and print the performance
    double secondsTotal = 0.001 * (double)msecTotal;
    double multsPerSecond = (float)SIZE / secondsTotal;
    double megaMultsPerSecond = multsPerSecond / 1000000;
    printf( "Array Size = %10d, MegaCalculatiosn / Second = %10.2lf\n", SIZE, megaMultsPerSecond );

    // Copy result from device to host
    status = hipMemcpy( hSum, dSum, (SIZE / BLOCKSIZE) * sizeof( float ), hipMemcpyDeviceToHost );
        checkCudaErrors( status );

    // Check the probability:
    double sum = 0.;
    for( int i = 0; i < SIZE / BLOCKSIZE; i++ )
    {
        sum += (double)hSum[ i ];
    }
    double prob = sum / (double)SIZE;
    printf( "Probability = %10.2lf\n", prob );

    // Clean up memory
    delete[] hXCS;
    delete[] hYCS;
    delete[] hRS;
    delete[] hSum;

    status = hipFree( dXCS );
        checkCudaErrors( status );
    status = hipFree( dYCS );
        checkCudaErrors( status );
    status = hipFree( dRS  );
        checkCudaErrors( status );
    status = hipFree( dSum );
        checkCudaErrors( status );
    
    return 0;
}

float
Ranf( float low, float high )
{
    float r = (float) rand();           // 0 - RAND_MAX
    float t = r / (float) RAND_MAX;     // 0. - 1.

    return low + t * ( high - low );
}

int
Ranf( int ilow, int ihigh )
{
    float low = (float)ilow;
    float high = ceil( (float)ihigh );

    return (int) Ranf( low, high );
}

void
TimeOfDaySeed()
{
    struct tm y2k = { 0 };
    y2k.tm_hour = 0;
    y2k.tm_min  = 0;
    y2k.tm_sec  = 0;
    y2k.tm_year = 100;
    y2k.tm_mon  = 0;
    y2k.tm_mday = 1;

    time_t timer;
    time( &timer );
    double seconds = difftime( timer, mktime( &y2k ) );
    unsigned int seed = (unsigned int)( 1000. * seconds );  // milliseconds
    srand( seed );
}
